
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <malloc.h>
 
#ifndef DATA_TYPE
#define DATA_TYPE float
#endif

#define CHECK(call)                                           \
{                                                             \
const hipError_t error = call;                               \
if (error != hipSuccess)                                     \
{                                                             \
fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);        \
fprintf(stderr, "code: %d, reason: %s\n", error,              \
cudaGetErrorString(error));                                   \
}                                                             \
}

__device__ int getGlobalIdx_1D_1D() {
    // Operações -> multiply: 1 add: 1 (2 FLOPs).
    // printf("getGlobalIdx_1D_1D.\n");
    return blockIdx.x * blockDim.x + threadIdx.x;
}
__device__ int getGlobalIdx_1D_2D() {
    // Operações -> multiply: 3 add: 2 (5 FLOPs).
    // printf("getGlobalIdx_1D_2D.\n");
    return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x
            + threadIdx.x;
}
__device__ int getGlobalIdx_1D_3D() {
    // Operações -> multiply: 6 add: 3 (9 FLOPs).
    // printf("getGlobalIdx_1D_3D.\n");
    return blockIdx.x * blockDim.x * blockDim.y * blockDim.z
            + threadIdx.z * blockDim.y * blockDim.x + threadIdx.y * blockDim.x
            + threadIdx.x;
}
__device__ int getGlobalIdx_2D_1D() {
    // Operações -> multiply: 2 add: 2 (4 FLOPs).
    // printf("getGlobalIdx_2D_1D.\n");
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_2D_2D() {
    // Operações -> multiply: 4 add: 3 (7 FLOPs).
    // printf("getGlobalIdx_2D_2D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y)
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_2D_3D() {
    // Operações -> multiply: 7 add: 4 (11 FLOPs).
    // printf("getGlobalIdx_2D_3D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
    + (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x)
            + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_3D_1D() {
    // Operações -> multiply: 4 add: 3 (7 FLOPs).
    // printf("getGlobalIdx_3D_1D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * blockDim.x + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_3D_2D() {
    // Operações -> multiply: 6 add: 4 (10 FLOPs).
    // printf("getGlobalIdx_3D_2D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y)
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}
__device__ int getGlobalIdx_3D_3D() {
    // Operações -> multiply: 9 add: 5 (14 FLOPs).
    // printf("getGlobalIdx_3D_3D.\n");
    int blockId = blockIdx.x + blockIdx.y * gridDim.x
            + gridDim.x * gridDim.y * blockIdx.z;
    int threadId = blockId * (blockDim.x * blockDim.y * blockDim.z)
            + (threadIdx.z * (blockDim.x * blockDim.y))
            + (threadIdx.y * blockDim.x) + threadIdx.x;
    return threadId;
}

/* Tipo para o ponteiro de função. */
typedef int (*op_func) (void);

/* Tabela de funções para chamada parametrizada. */
__device__ op_func getGlobalIdFunc[9] = { getGlobalIdx_1D_1D, getGlobalIdx_1D_2D, getGlobalIdx_1D_3D, 
                      getGlobalIdx_2D_1D, getGlobalIdx_2D_2D, getGlobalIdx_2D_3D,
                      getGlobalIdx_3D_1D, getGlobalIdx_3D_2D, getGlobalIdx_3D_3D};

void init_arrays(DATA_TYPE* a, DATA_TYPE* b, int n){
    int i;
    // double invrmax = 1.0 / RAND_MAX;
    for (i = 0; i < n; i++) {
        // a[i] = rand() * invrmax;
        // b[i] = rand() * invrmax;
        a[i] = 0.5;
        b[i] = 0.5;
    }
}

__global__ void vecAdd(DATA_TYPE *a, DATA_TYPE *b, DATA_TYPE *c, int n, int funcId){
    //Thread ID
    int id = getGlobalIdFunc[funcId]();
    printf("id: %d\n", id);
    if (id < n)
        c[id] = a[id] + b[id];
}
 
int main(int argc, char **argv){
    int i;
    hipError_t err;
    int n = 0;
    int kernel = 0;
    int funcId = 0;
    int gpuId = 0;
 	
 	/*if (argc != 11) {
        printf("Uso: %s <kernel> <g.x> <g.y> <g.z> <b.x> <b.y> <b.z> <nx> <funcId> <gpuId>\n", argv[0]);
        printf("     funcId:\n");
        printf("     0: 1D_1D, 1: 1D_2D, 2: 1D_3D\n");
        printf("     3: 2D_1D, 4: 2D_2D, 5: 2D_3D\n");
        printf("     6: 3D_1D, 7: 3D_2D, 8: 3D_3D\n");
        return 0;
    }
    else{
        printf("#argumentos (argc): %d\n", argc);
        for (i = 0; i < argc; ++i) {
           printf(" argv[%d]: %s\n", i, argv[i]);
        }*/
    
        kernel = 0;
        n = 1024;
        funcId = 4;
        gpuId = 0;
        //printf("Executando: %s sumvector_kernel_%d grid(%d, %d, %d) block(%d, %d, %d) %d\n", argv[0], kernel, atoi(argv[2]), atoi(argv[3]), atoi(argv[4]), atoi(argv[5]), atoi(argv[6]), atoi(argv[7]), n);
    //}
  
    /* Recuperar as informações da GPU. */
    printf("%s Starting...\n", argv[0]);
  
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
  
    if (deviceCount == 0) {
        printf("Não existem dispositivos com suporte a CUDA.\n");
        return 0;
    } else {
        printf("Existem %d dispositivos com suporte a CUDA.\n", deviceCount);
        if(gpuId > (deviceCount - 1)){
            printf("Não existe um dispositivo sob o id: %d. Utilize %d a %d\n", gpuId, 0, (deviceCount - 1));
            return 0;
        }
    }
    /* Define the gpu id to work */
    hipSetDevice(gpuId);
     
    /* Alocação das estruturas. */
    // Size, in bytes, of each vector
    size_t bytes = sizeof(DATA_TYPE) * n;
    printf(" sizeof(DATA_TYPE): %d\n", (int) sizeof(DATA_TYPE));
    size_t totalmem = (3 * bytes);
    printf(" Qtd bytes por estrutura: %zu total: %zu\n", bytes, totalmem);
  
    /* Dados no host. */
    printf("Allocate memory for each vector on host.\n");
    DATA_TYPE *h_a = (DATA_TYPE*) malloc(bytes);
    DATA_TYPE *h_b = (DATA_TYPE*) malloc(bytes);
    DATA_TYPE *h_c = (DATA_TYPE*) malloc(bytes);
  
    /* Dados no dispositivo. */
    DATA_TYPE *d_a;
    DATA_TYPE *d_b;
    // Device output vector.
    DATA_TYPE *d_c;
 
    printf("Allocate memory for each vector on GPU.\n");
    // Allocate memory for each vector on GPU
    hipMalloc(&d_a, bytes);
    hipMalloc(&d_b, bytes);
    hipMalloc(&d_c, bytes);
 
    init_arrays(h_a, h_b, n);
  
    printf("Copy host vectors to device.\n");
    // Copy host vectors to device
    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, bytes, hipMemcpyHostToDevice);

    /* Definição do arranjo de threads em blocos do grid. */
    // (   1,   2,   8,   1,   2,  32 )
    dim3 grid(1, 2, 8);
    dim3 block(1, 2, 32);

    printf("Execute the kernel.\n");
    hipEvent_t start_event, stop_event;
    float time_kernel_execution;
    int eventflags = hipEventBlockingSync;
    hipEventCreateWithFlags(&start_event, eventflags);
    hipEventCreateWithFlags(&stop_event, eventflags);

    /* O kernel e a função de calculo do id global são escolhidos conforme o parâmetros.*/
    switch (kernel){
        case 0:
            printf("Executing sincos_kernel_%d.\n", kernel);
            vecAdd<<<grid, block>>>(d_a, d_b, d_c, n, funcId);
        break;
        default :
            printf("Invalid kernel number.\n");
    }

    err = hipGetLastError();
  
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch kernel (error code %s)!\n",
        hipGetErrorString(err));
        exit (EXIT_FAILURE);
    }
    /* Synchronize */
    hipDeviceSynchronize();
  
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&time_kernel_execution, start_event, stop_event);
    printf("Time Kernel Execution: %f s\n", (time_kernel_execution / 1000.0f));
    printf("Time Kernel Execution: %f ms\n", (time_kernel_execution));
  
    printf("Copy array back to host.\n");
    // Copy array back to host
    hipMemcpy(h_c, d_c, bytes, hipMemcpyDeviceToHost);
    // Release device memory
  
    printf("Liberando as estruturas alocadas na Memória da GPU.\n");
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    DATA_TYPE soma,media;
    soma = 0;
    for(i=0; i<n; i++){
        soma = soma + h_c[i];
        
        //printf("resultado: %f\n", h_c[n]);
    }
    media = soma / n;
    printf("resultado: %f\n", media);
    printf("Liberando as estruturas alocadas na Memória do host.\n");
    free(h_a);
    free(h_b);
    free(h_c);
  
    printf("Reset no dispositivo.\n");
    CHECK(hipDeviceReset());
  
    printf("Done.\n");
  
    return 0;
}